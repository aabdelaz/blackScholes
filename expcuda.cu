#include "hip/hip_runtime.h"
// The caller of this code is responsible for calling culaInitialize and culaShutdown
#define POL 6
#define IDX(i, j, n) n*(j) + i 

#include <math.h>
#include <stdlib.h>
#include <string.h> 
#include <stdio.h>
#include <float.h>

#include "common.h"

void pm(float *A, int n) {
    int i, j;
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) 
            printf("%f ", A[i + j*n]);
        printf("\n");
    }
    printf("\n");
}

void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS error: Status code %d\n", status);
        exit(1);
    }
}

void checkError(hipError_t error) {
    if (error != hipSuccess) {
        fprintf(stderr, "!!!! CUDA error: Error code %d\n", error);
        exit(1);
    }
}

void checkCulaStatus(culaStatus status)
{
    char buf[256];
	
    if(!status)
        return;
		printf("nooooo %d\n", status);

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    culaShutdown();
    exit(EXIT_FAILURE);
}

float infinity_norm(float *A, int n) {
    int i, j;
    float sum, max = FLT_MIN, ent;
    for (i = 0; i < n; i++) {
		sum = 0;
		for (j = 0; j < n; j++) {
            ent = A[IDX(i, j, n)];
            if (ent < 0) ent *= -1;
	        sum += ent;
		}

			if (sum > max) max = sum;
    }
			
    return max;
}

// a is the matrix you have, e is the one you'll fill in
// now I'm going to use cublas, assume A and E are device pointers
void padeExp(hipblasHandle_t handle, float *A, float *E, int n) {
    culaStatus status;
    hipblasStatus_t bS;
    hipError_t error;
    float s, s_factor, one = 1, zero = 0, minus = -1, two = 2, m_two = -2;
    float *Q, *A2, *P;
    int i, j, f, *piv, scaled = 0;
    float c[POL + 1];
    float *A_h = (float*)malloc(n*n*sizeof(A_h[0]));

    c[0] = 1;
    for (i = 0; i < POL; i++) {
      	c[i + 1] = c[i]*((double)(POL - i)/((i + 1)*(2*POL - i)));
    } 
 
    int blockSize = 16;
    dim3 threads(blockSize, blockSize);
    dim3 grid((n + blockSize - 1)/blockSize, (n + blockSize - 1)/blockSize);

    // so I'll put this in for now to see if it's faster 
    int threadsPerBlock = blockSize*blockSize;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    extern float *p; 
/*    addRows<<<blocksPerGrid, threadsPerBlock>>>(A, p, n);
 
    int idx;
    bS = hipblasIsamax(handle, n, p, 1, &idx);
    checkCublasStatus(bS);

    // this will put s_factor in p[0] and 1/s_factor in p[1]
    findScale<<<1, 1>>>(p, idx);

    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    bS = hipblasSscal(handle, n*n, p, A, 1); 
    checkCublasStatus(bS); 
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
*/

    bS = hipblasGetVector(n*n, sizeof(A[0]), A, 1, A_h, 1);
    checkCublasStatus(bS);

    // scale here
    s = infinity_norm(A_h, n);
    if (s > 0.5) {
        scaled = 1;
        f = (int) (log(s)/log(2));
        s = MAX(0,f + 2);
        s_factor = pow(2, -s);
        bS = hipblasSscal(handle, n*n, &s_factor, A, 1);
        checkCublasStatus(bS);
    }

    error = hipMalloc((void**)&piv, n*n*sizeof(Q[0]));
    checkError(error);

    error = hipMalloc((void**)&Q, n*n*sizeof(Q[0]));
    checkError(error);

    error = hipMalloc((void**)&P, n*n*sizeof(P[0]));
    checkError(error);

    error = hipMemset((void*)Q, 0, n*n*sizeof(Q[0]));
    checkError(error);

    error = hipMemset((void*)P, 0, n*n*sizeof(P[0]));
    checkError(error);

    // allocate space for A2; no need to initialize memory

    error = hipMalloc((void**)&A2, n*n*sizeof(A2[0]));
    checkError(error);

    bS = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, A, n, A, n, &zero, A2, n);
    checkCublasStatus(bS);

    addDiag<<<grid, threads>>>(Q, n, c[POL]);
    addDiag<<<grid, threads>>>(P, n, c[POL - 1]);

    int odd = 1;
    for (i = POL - 2; i >= 0; i--) {
        if (odd == 1) {
            // Q = Q*A2
            bS = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, Q, n, A2, n, &zero, Q, n);            
            checkCublasStatus(bS);

            // Q = Q + c[k]*I
            addDiag<<<grid, threads>>>(Q, n, c[i]);

        }
        else {
            // P = P*A2 
            bS = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, P, n, A2, n, &zero, P, n);            
            checkCublasStatus(bS);

            // P = P + c[k]*I
            addDiag<<<grid, threads>>>(P, n, c[i]);

        }
        odd = 1-odd;
  	}  

  	if (odd == 1) {
    	// Q = Q*A
        bS = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, Q, n, A, n, &zero, Q, n);            
        checkCublasStatus(bS);
  	}
  	else {
    	// P = P*A
        bS = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, P, n, A, n, &zero, P, n);            
        checkCublasStatus(bS);
  	}
  
  	// Q = Q - P
    bS = hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &one, Q, n, &minus, P, n, Q, n);
    checkCublasStatus(bS);            

  	// Find X s.t. QX = Pi
    error = hipMemset((void*)piv, 0, n*sizeof(int));
    checkError(error);

  	status = culaDeviceSgesv(n, n, Q, n, piv, P, n);
  	checkCulaStatus(status);

 	// now P = X
 
  	if (odd == 0) {
        // E = 2*P
         bS = hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &two, P, n, &zero, NULL, n, E, n);
         checkCublasStatus(bS);
         addDiag<<<grid, threads>>>(E, n, 1);
    }            
  	else {
        bS = hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &m_two, P, n, &zero, NULL, n, E, n);
        checkCublasStatus(bS);
        addDiag<<<grid, threads>>>(E, n, -1);
    }

	for (i = 0; i < s; i++) {
        bS = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &one, E, n, E, n, &zero, E, n);            
        checkCublasStatus(bS);
    }
/*
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    bS = hipblasSscal(handle, n*n, p + 1, A, 1); 
    checkCublasStatus(bS);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
*/

    if (scaled == 1) {
        s_factor = 1./s_factor;
        bS = hipblasSscal(handle, n*n, &s_factor, A, 1);
        checkCublasStatus(bS);
    }

    free(A_h);


    error = hipFree(Q);
    checkError(error);

    error = hipFree(P);
    checkError(error);

    error = hipFree(A2);
    checkError(error);

    return;
}

void phi(hipblasHandle_t handle, float *A, float *E, int n) {
    float one = 1, minus = -1;
    int i, *piv;
    hipblasStatus_t bS;
    hipError_t error;
    culaStatus status;
 
    // we want AX = e^A - I

    padeExp(handle, A, E, n);

    int blockSize = 16;
    dim3 threads(blockSize, blockSize);
    dim3 grid((n + blockSize - 1)/blockSize, (n + blockSize - 1)/blockSize);

    addDiag<<<grid, threads>>>(E, n, -1);

    // now E = e^A - I

  	// Find X s.t. AX = E
  	error = hipMalloc((void**)&piv, n*sizeof(int));
    checkError(error);

    error = hipMemset((void*)piv, 0, n*sizeof(int));
    checkError(error);

    status = culaDeviceSgesv(n, n, A, n, piv, E, n);
  	checkCulaStatus(status);

  	// now E = X

    // cleanup
    error = hipFree(piv);
    checkError(error);

}

/*
int main(void) {
    #define N 5
    hipblasStatus_t status;
    hipblasHandle_t handle;
    hipError_t error;
    float A_h[N*N] = { -0.16580, 0.22570, 0.00000, 0.00000, 0.00000, 0.25460,
  -0.73720,
   0.29330,
   0.00000,
   0.00000,
  -0.00220,
   0.33500,
  -0.53390,
   0.20670,
   0.00000,
  -0.002100,
  -0.0019000,
   0.2369000,
  -0.3663000,
   0.1378000,
  -0.0019000,
  -0.0018000,
  -0.0025000,
   0.1566000,
  -0.2340000
};

    float E_h[N*N];
    int i, j;
    //float A[4] = {1, 0, 0, 1};
    float *E, *A;

  	error = hipMalloc((void**)&E, N*N*sizeof(float));
    checkError(error);

  	error = hipMalloc((void**)&A, N*N*sizeof(float));
    checkError(error);

    status = hipblasCreate(&handle);
    checkCublasStatus(status);
    culaInitialize();

    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) 
            printf("%f ", A_h[i + j*N]);
        printf("\n");
    }
    printf("\n");

    error = hipMemcpy((void*)A, (void*)A_h, N*N*sizeof(float), hipMemcpyHostToDevice);
    checkError(error); 

    //padeExp(handle, A, E, N);
    phi(handle, A, E, N);

    error = hipMemcpy((void*)E_h, (void*)E, N*N*sizeof(float), hipMemcpyDeviceToHost);
    checkError(error); 

    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) 
            printf("%f ", E_h[i + j*N]);
        printf("\n");
    }

    status = hipblasDestroy(handle);
    checkCublasStatus(status);

    error = hipFree(E);
    checkError(error);

    error = hipFree(A);
    checkError(error);


    culaShutdown();
}*/ 
